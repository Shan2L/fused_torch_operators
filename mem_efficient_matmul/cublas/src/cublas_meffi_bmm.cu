#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "errors.h"

__global__ void check_dev_data(const int id, float* data, int pitch, int sizex)
{
    printf("id: %d, batch: %d, y: %d, x: %d, data: %f\n", 
    id, blockIdx.x, threadIdx.y, threadIdx.x, 
    data[(blockIdx.x * blockDim.y + threadIdx.y) * pitch/sizeof(float) + threadIdx.x]);
}

#define debug_name(name, var) \
    std::cout << name <<": " << var <<std::endl;

torch::Tensor cublas_meffi_bmm(torch::Tensor& mat_a, torch::Tensor& mat_b,
                                     int tile_size) {
    std::vector<int64_t> a_size = mat_a.sizes().vec();
    std::vector<int64_t> b_size = mat_b.sizes().vec();
    std::vector<int64_t> a_stride = mat_a.strides().vec();
    std::vector<int64_t> b_stride = mat_b.strides().vec();
    TORCH_INTERNAL_ASSERT(a_size.size() == b_size.size(), "Dimension of two input must be same now.");
    int ndim = a_size.size();

    // TODO dimension check and broadcast

    TORCH_INTERNAL_ASSERT(a_size[ndim-1] == b_size[ndim-2],
                          "The dimensions of mat1 and mat2 do not match.");
    TORCH_INTERNAL_ASSERT(mat_a.scalar_type() == mat_b.scalar_type(),
                          "Datatype of matrix a and matrix b should be same!");




    // int64_t batch_size = std::accumulate(a_size.rbegin()+2, a_size.rend(), 1, std::multiplies<int64_t>());
    int64_t batch_size = a_size[0];
    int64_t m = a_size[ndim-2];
    int64_t n = b_size[ndim-1];
    int64_t k = a_size[ndim-1];

    debug_name("batch_size", batch_size)
    debug_name("m", m)
    debug_name("n", n)
    debug_name("k", k)


    torch::Tensor out = at::zeros({batch_size, m, n}, mat_a.options());

    void* out_ptr = out.data_ptr();
    void* a_ptr = mat_a.data_ptr();
    void* b_ptr = mat_b.data_ptr();

    // TODO Asynchronization implementation.
    // TODO replace the resource creation with pytorch builtin function.

    // hipStream_t stream_a, stream_b, stream_out2dev, stream_cublas,
    //     stream_out2host;
    // hipEvent_t event_a, event_b, event_out2dev, event_cublas, event_out2host;
    // CUDA_CHECK(hipStreamCreate(&stream_a));
    // CUDA_CHECK(hipStreamCreate(&stream_b));
    // CUDA_CHECK(hipStreamCreate(&stream_out2dev));
    // CUDA_CHECK(hipStreamCreate(&stream_cublas));
    // CUDA_CHECK(hipStreamCreate(&stream_out2host));
    // CUDA_CHECK(hipEventCreate(&event_a));
    // CUDA_CHECK(hipEventCreate(&event_b));
    // CUDA_CHECK(hipEventCreate(&event_out2dev));
    // CUDA_CHECK(hipEventCreate(&event_cublas));
    // CUDA_CHECK(hipEventCreate(&event_out2host));

    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    // CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream_cublas));

    // TODO replace the hipMalloc with pytorch inplace function.
    // 目标内存指针设置
    hipMemcpy3DParms copyParams_a = {0};
    hipPitchedPtr dstPtr_a;
    CUDA_CHECK(hipMalloc3D(&dstPtr_a, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
    hipPitchedPtr srcPtr_a = make_hipPitchedPtr((void*)a_ptr, k*sizeof(float), k, m);
    copyParams_a.srcPtr = srcPtr_a;
    copyParams_a.dstPtr = dstPtr_a;
    copyParams_a.kind = hipMemcpyHostToDevice;
    copyParams_a.dstPos = make_hipPos(0, 0, 0);

    debug_name("dstPtr_a.pitch", dstPtr_a.pitch)
    debug_name("dstPtr_a.xsize", dstPtr_a.xsize)
    debug_name("dstPtr_a.ysize", dstPtr_a.ysize)

    hipMemcpy3DParms copyParams_b = {0};
    hipPitchedPtr dstPtr_b;
    CUDA_CHECK(hipMalloc3D(&dstPtr_b, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
    hipPitchedPtr srcPtr_b = make_hipPitchedPtr((void*)b_ptr, n*sizeof(float), n, k);
    copyParams_b.srcPtr = srcPtr_b;
    copyParams_b.dstPtr = dstPtr_b;
    copyParams_b.kind = hipMemcpyHostToDevice;
    copyParams_b.dstPos = make_hipPos(0, 0, 0);


    hipPitchedPtr dstPtr_out;
    CUDA_CHECK(hipMalloc3D(&dstPtr_out, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
    hipPitchedPtr srcPtr_out = make_hipPitchedPtr((void*)out_ptr, n*sizeof(float), n, m);
    
    hipMemcpy3DParms copyParams_out2dev = {0};
    copyParams_out2dev.srcPtr = srcPtr_out;
    copyParams_out2dev.dstPtr = dstPtr_out;
    copyParams_out2dev.kind = hipMemcpyHostToDevice;
    copyParams_out2dev.dstPos = make_hipPos(0, 0, 0);

    hipMemcpy3DParms copyParams_out2host = {0};
    copyParams_out2host.srcPtr = dstPtr_out;
    copyParams_out2host.dstPtr = srcPtr_out;
    copyParams_out2host.kind = hipMemcpyDeviceToHost;
    copyParams_out2host.srcPos = make_hipPos(0, 0, 0);

    float alpha = 1.0;
    float beta = 1.0;

    size_t tile_num_m = (m + tile_size -1) / tile_size;
    size_t tile_num_n = (n + tile_size -1) / tile_size;
    size_t tile_num_k = (k + tile_size -1) / tile_size;

    debug_name("tile_num_m", tile_num_m)
    debug_name("tile_num_n", tile_num_n)
    debug_name("tile_num_k", tile_num_k)


    // TODO the asynchonization has not been enable correctly, the performance looks the same as the serial execution.
    for (int m_tile_idx = 0; m_tile_idx < tile_num_m; m_tile_idx++) {
        for (int n_tile_idx = 0; n_tile_idx < tile_num_n; n_tile_idx++) {
            for (int k_tile_idx = 0; k_tile_idx < tile_num_k; k_tile_idx++) {
                std::cout << m_tile_idx << ", " << n_tile_idx << ", " << k_tile_idx<< std::endl;
                size_t element_m = m % tile_size == 0 ? tile_size : (m_tile_idx == tile_num_m - 1 ? (m % tile_size) : tile_size);
                size_t element_n = n % tile_size == 0 ? tile_size : (n_tile_idx == tile_num_n - 1 ? (n % tile_size) : tile_size);
                size_t element_k = k % tile_size == 0 ? tile_size : (k_tile_idx == tile_num_k - 1 ? (k % tile_size) : tile_size);

                debug_name("element_m", element_m)
                debug_name("element_n", element_n)
                debug_name("element_k", element_k)
                // copy a tile of A to GPU

                if (element_m != tile_size || element_k != tile_size)
                {
                    CUDA_CHECK(hipMemset3D(dstPtr_a, 0, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
                }
                copyParams_a.extent = make_hipExtent(element_k*sizeof(float), element_m, batch_size);
                copyParams_a.srcPos = make_hipPos(k_tile_idx*tile_size*sizeof(float), m_tile_idx*tile_size, 0);
                CUDA_CHECK(hipMemcpy3D(&copyParams_a));
                hipDeviceSynchronize();
                check_dev_data<<<batch_size, dim3(element_k, element_m)>>>(1, (float*)dstPtr_a.ptr, dstPtr_a.pitch, dstPtr_a.xsize);

                // copy a tile of B to GPU
                if (element_n != tile_size || element_k != tile_size)
                {
                    CUDA_CHECK(hipMemset3D(dstPtr_b, 0, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
                }
                copyParams_b.extent = make_hipExtent(element_n* sizeof(float), element_k, batch_size);
                copyParams_b.srcPos = make_hipPos(n_tile_idx*tile_size*sizeof(float), k_tile_idx*tile_size, 0);
                CUDA_CHECK(hipMemcpy3D(&copyParams_b));
                check_dev_data<<<batch_size, dim3(element_n, element_k)>>>(2, (float*)dstPtr_b.ptr, dstPtr_b.pitch, dstPtr_a.xsize);

                // copy a tile of C to GPU
                if (element_m != tile_size || element_n != tile_size)
                {
                    CUDA_CHECK(hipMemset3D(dstPtr_out, 0, make_hipExtent(tile_size*sizeof(float), tile_size, batch_size)));
                }
                copyParams_out2dev.extent = make_hipExtent(element_n* sizeof(float), element_m, batch_size);
                copyParams_out2dev.srcPos = make_hipPos(n_tile_idx*tile_size*sizeof(float), m_tile_idx*tile_size, 0);
                CUDA_CHECK(hipMemcpy3D(&copyParams_out2dev));
                check_dev_data<<<batch_size, dim3(element_m, element_n)>>>(4, (float*)dstPtr_out.ptr, dstPtr_out.pitch, dstPtr_out.xsize);

                // invoke hipblasSgemm C=A@B+C


                CUBLAS_CHECK(
                    hipblasSgemmStridedBatched(
                    cublas_handle, 
                    HIPBLAS_OP_N, 
                    HIPBLAS_OP_N, 
                    tile_size, tile_size, tile_size, 
                    &alpha, 
                    (float*)dstPtr_b.ptr, dstPtr_b.pitch/sizeof(float), dstPtr_b.pitch/sizeof(float)*tile_size, 
                    (float*)dstPtr_a.ptr, dstPtr_a.pitch/sizeof(float), dstPtr_a.pitch/sizeof(float)*tile_size, 
                    &beta, 
                    (float*)dstPtr_out.ptr, dstPtr_out.pitch/sizeof(float), dstPtr_out.pitch/sizeof(float)*tile_size,
                    batch_size));

                check_dev_data<<<batch_size, dim3(element_m, element_n)>>>(3, (float*)dstPtr_out.ptr, dstPtr_out.pitch, dstPtr_out.xsize);


                // CUDA_CHECK(hipEventRecord(event_cublas, stream_cublas));
                hipDeviceSynchronize();
                // copy C to CPU


                copyParams_out2host.dstPos = make_hipPos(n_tile_idx*tile_size*sizeof(float), m_tile_idx*tile_size, 0);
                std::cout << "cudapos: " << "x: "<<n_tile_idx*tile_size << ", y: "<<m_tile_idx*tile_size, 
                copyParams_out2host.extent = make_hipExtent(element_n* sizeof(float), element_m, batch_size);
                CUDA_CHECK(hipMemcpy3D(&copyParams_out2host));

                std::cout << "output host: "<<std::endl;
                std::cout << "==============================================="<< std::endl;
                for (int b=0; b<batch_size; b++)
                {
                    for (int i=0; i<m; i++)
                    {
                        for(int j=0; j<n; j++)
                        {
                            std::cout << ((float*)out_ptr)[b*m*n+i*n+j] << ", ";
                        }   
                        std::cout << std::endl;
                    }
                    std::cout <<std::endl;
                }

                std::cout << "==============================================="<< std::endl;
            }
        }
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // CUDA_CHECK(hipFree(a_dev));
    // CUDA_CHECK(hipFree(b_dev));
    // CUDA_CHECK(hipFree(out_dev));   
    
    return out;
}

